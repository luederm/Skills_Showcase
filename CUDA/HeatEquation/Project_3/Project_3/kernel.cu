#include "hip/hip_runtime.h"
/*
	@author Matthew Lueder
	@description Apply heat equation to arrays of varying sizes and dimensionality with CUDA
*/

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>  
#include <boost\timer.hpp>
#include "CImg.h"

using namespace cimg_library;

/* CONTROLS / SWITCHS */

// So we can experiment with differnt precision
typedef float floatp;

// For all implementations
#define ITERATIONS 100000 
#define H_HEAT 100.00
#define H_ROOM_TEMP 23.00
__constant__ floatp HEAT = H_HEAT;
__constant__ floatp ROOM_TEMP = H_ROOM_TEMP;

// For 1-D multi-block implementation
#define H_NUM_SLICES 1000000
__constant__ size_t D_NUM_SLICES = H_NUM_SLICES;

// For 2-D implementation
#define H_ROOM_X 1000
#define H_ROOM_Y 1000
#define STEPS 1
#define DRAW_IMG false
__constant__ int ROOM_X = H_ROOM_X;
__constant__ int ROOM_Y = H_ROOM_Y;

// Macros
#define ROOM_INDEX(_x, _y) (_x) + ROOM_X * (_y)
#define GLOBAL_INDEX(_x,_y) (_y) * (gridDim.x * blockDim.x) + (_x)
#define LOCAL_INDEX(_x,_y) (_y) * blockDim.x + (_x)
#define CHECK(cudaStatus) if(cudaStatus != hipSuccess) printf("%d> CUDA ERROR: %s\n", __LINE__, hipGetErrorString(cudaStatus))

/*
	This kernel initializes rods in device memory
*/
__global__ void  initialize(floatp *rod)
{
	// Get unique id
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	// Set first element to HEAT, and all others to ROOM_TEMP
	if (id < D_NUM_SLICES && id != 0)
	{
		rod[id] = ROOM_TEMP;
	}
	else if (id == 0)
	{
		rod[0] = HEAT;
	}
}

/*
This kernel initializes rooms in device memory
*/
__global__ void  initialize2D(floatp* room)
{
	// Get x and y value
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	// Set first element to HEAT, and all others to ROOM_TEMP
	if (x < ROOM_X && y < ROOM_Y)
	{
		int room_index = ROOM_INDEX(x, y);

		if (room_index == 0)
		{
			room[0] = HEAT;
		}
		else
		{
			room[room_index] = ROOM_TEMP;
		}
	}
}

/*
	Multi-block solution which uses a combination of shared and global memory. 
	Can handle larger input arrays.
	Running this kernel once is the equivalent of going foward one time step.
*/
__global__ void applyHeat(floatp* rod_in, floatp* rod_out)
{
	extern __shared__ floatp sharedMem[];

	// Get unique id
	int uid = (blockIdx.x * blockDim.x + threadIdx.x) + 1;

	// Thread id
	int tid = threadIdx.x;

	// Load global data to shared data
	if (uid < D_NUM_SLICES)
		sharedMem[tid] = rod_in[uid];

	__syncthreads();

	if (uid < (D_NUM_SLICES-1))
	{ 
		if (tid == 0)
		{
			rod_out[uid] = (rod_in[uid-1] + sharedMem[tid+1]) / 2;
		}
		else if (tid == blockDim.x - 1)
		{
			rod_out[uid] = (rod_in[uid+1] + sharedMem[tid-1]) / 2;
		}
		else
		{
			rod_out[uid] = (sharedMem[tid-1] + sharedMem[tid+1]) / 2;
		}
	}
	else if (uid == (D_NUM_SLICES - 1))
	{
		if (tid != 0)
		{
			rod_out[uid] = (sharedMem[tid-1] + sharedMem[tid]) / 2;
		}
		else
		{
			rod_out[uid] = (rod_in[uid-1] + sharedMem[tid]) / 2;
		}
	}
}


/*
	1-block solution which only uses shared memory and executes multiple time steps in one kernel call.
	This solution is limited to arrays of size 1024 / 1024 threads per block.
	No need to initialize (done in this kernel).
	Super fast but very limited.
*/
__global__ void applyHeat(floatp* rod)
{
	 __shared__ floatp sMem[1025];
	 __shared__ floatp sMem2[1025];

	// Thread id (1-1024)
	int id = threadIdx.x + 1;

	// Set heat source
	if (id == 1)
	{ 
		sMem[0] = HEAT;
		sMem2[0] = HEAT;
	}

	// Set elements to room temp
	sMem[id] = ROOM_TEMP;

	__syncthreads();

	int iteration = 0;
	while (iteration < ITERATIONS)
	{ 
		if (iteration % 2 == 0)
		{ 
			if (id != 1024)
			{
				sMem2[id] = (sMem[id - 1] + sMem[id + 1]) / 2;
			}
			else
			{
				sMem2[id] = (sMem[id - 1] + sMem[id]) / 2;
			}
		}
		else
		{
			if (id != 1024)
			{
				sMem[id] = (sMem2[id - 1] + sMem2[id + 1]) / 2;
			}
			else
			{
				sMem[id] = (sMem2[id - 1] + sMem2[id]) / 2;
			}
		}
		
		__syncthreads();
		++iteration;
	}

	if (iteration % 2 == 0)
	{ 
		rod[id - 1] = sMem[id];
	}
	else
	{
		rod[id - 1] = sMem2[id];
	}
}


/*
	Multi-block solution which uses a combination of shared and global memory.
	Running this kernel once is the equivalent of going foward one time step.
	Use square blocks (32x32) only.
*/
__global__ void applyHeat2D(floatp* room_in, floatp* room_out)
{
	__shared__ floatp sMem[1024];

	// Get X and Y relative to global block -> use to calculate room index
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int pos = ROOM_INDEX(x,y);

	// Unique id for threads in block
	int t_x = threadIdx.x;
	int t_y = threadIdx.y;
	int t_pos = LOCAL_INDEX(t_x, t_y);

	__syncthreads();

	// Ignore threads outside of room boundaries
	if (x < ROOM_X && y < ROOM_Y)
	{ 
		// Load global data to shared data
		sMem[t_pos] = room_in[pos];

		__syncthreads();

		// Elements needed to perform calculation
		floatp top, bottom, left, right;
		
		// ~ Find top element ~
		if (t_y == 0)
		{
			if (y == 0)
			{
				// top element is a wall of the room (re-use this element's data)
				top = sMem[t_pos];
			}
			else
			{
				// top element in room but beyond block boundry
				top = room_in[ROOM_INDEX(x, y-1)];
			}
		}
		else
		{
			// top element within room and block
			top = sMem[LOCAL_INDEX(t_x, t_y-1)];
		}

		// ~ Find bottom element ~
		if (y == ROOM_Y - 1)
		{
			// Bottom element is the wall of the room
			bottom = sMem[t_pos];
		}
		else if (t_y == blockDim.y - 1)
		{
			// Bottom element is in room but beyond block boundry
			bottom = room_in[ROOM_INDEX(x, y+1)];
		}
		else
		{
			// Bottom element in block and room
			bottom = sMem[LOCAL_INDEX(t_x, t_y+1)];
		}

		// ~ Find left element ~
		if (x == 0)
		{
			// Left element is the wall of room
			left = sMem[t_pos];
		}
		else if (t_x == 0)
		{
			// Left element beyond block boundry
			left = room_in[ROOM_INDEX(x-1, y)];
		}
		else
		{
			// Left element in block and room
			left = sMem[LOCAL_INDEX(t_x-1, t_y)];
		}

		// ~ Find right element ~
		if (x == ROOM_X - 1)
		{
			// Right element is wall of room
			right = sMem[t_pos];
		}
		else if (t_x == blockDim.x - 1)
		{
			// Right element is beyond block boundry
			right = room_in[ROOM_INDEX(x+1, y)];
		}
		else
		{
			// Right element is in block and room
			right = sMem[LOCAL_INDEX(t_x+1, t_y)];
		}

		// Now we calculate the new tempurature
		if (pos != 0)
			room_out[pos] = (top + bottom + right + left) / 4;
	}
}

void DrawImage(floatp* room, int height, int width, int num)
{
	CImg<unsigned char> img(width, height, 1, 3, 255);
	
	for (int i = 0; i < height; ++i)
	{
		for (int j = 0; j < width; ++j)
		{
			floatp value = room[j + i * width];

			// Get normalized data point [0,1]
			floatp scaledVal = (value - H_ROOM_TEMP) / (H_HEAT - H_ROOM_TEMP);

			// Calculate color
			unsigned char color[3];
			color[0] = scaledVal * 225;
			color[1] = 0;
			color[2] = 225 + scaledVal * -225;
			img.draw_point(j, i, color);
		}
	}

	img.save("images/heatmap.bmp", num);
}

void OneBlockHeat()
{
	hipError_t cudaStatus;
	const size_t SLICES = 1024;
	size_t size = SLICES * sizeof(floatp);

	// START TIMER
	boost::timer t;

	// Allocate space for return rod
	floatp* d_rod = NULL;
	cudaStatus = hipMalloc((void**)&d_rod, size);
	CHECK(cudaStatus);

	// # Threads/blocks always the same for this kernel
	int threadsPerBlock = 1024;
	int blocksPerGrid = 1;

	// Perform kernel operation
	applyHeat <<< blocksPerGrid, threadsPerBlock >>> (d_rod);
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);

	// PRINT TIME ELAPSED
	hipDeviceSynchronize();
	printf("Time elapsed: %f\n", t.elapsed());

	// Pull rod into host for testing
	floatp* h_rod = (floatp *)malloc(size);
	cudaStatus = hipMemcpy(h_rod, d_rod, size, hipMemcpyDeviceToHost);
	CHECK(cudaStatus);

	for (int i = 0; i <= 10; i += 1)
	{
		printf("%f\t", h_rod[i]);
	}
	printf("\n");
	printf("%f\n", h_rod[SLICES - 1]);

	int ind = 0;
	while (abs(h_rod[ind] - 23.00) > 0.0001)
	{
		++ind;
	}
	printf("first untouched = %d\n", ind);

	hipFree(d_rod);
	free(h_rod);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	CHECK(hipDeviceReset());
}

void MultiBlockHeat()
{
	hipError_t cudaStatus;
	size_t size = H_NUM_SLICES * sizeof(floatp);

	// START TIMER
	boost::timer t;

	// Allocate space for Rods
	floatp* d_rod = NULL;
	cudaStatus = hipMalloc((void**)&d_rod, size);
	CHECK(cudaStatus);

	floatp* d_rod2 = NULL;
	cudaStatus = hipMalloc((void**)&d_rod2, size);
	CHECK(cudaStatus);

	// Calculate threads/blocks for initialize kernel
	int threadsPerBlock = 1024;
	int blocksPerGrid = H_NUM_SLICES / threadsPerBlock;
	if (H_NUM_SLICES % threadsPerBlock != 0)
		++blocksPerGrid;

	// Perform initialize kernel
	printf("CUDA 'initialize' kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	initialize <<< blocksPerGrid, threadsPerBlock >>> (d_rod);
	initialize <<< blocksPerGrid, threadsPerBlock >>> (d_rod2);
	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);

	// Calculate blocks per grid for apply heat kernel
	blocksPerGrid = (H_NUM_SLICES - 1) / threadsPerBlock;
	if ((H_NUM_SLICES - 1) % threadsPerBlock != 0)
		++blocksPerGrid;

	// Perform applyHeat kernel for a specified number of iterations (time steps)
	printf("CUDA 'applyHeat' kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	int iteration = 0;
	while (iteration < ITERATIONS)
	{
		if (iteration % 2 == 0)
		{
			applyHeat <<< blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(floatp) >>> (d_rod, d_rod2);
		}
		else
		{
			applyHeat <<< blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(floatp) >>> (d_rod2, d_rod);
		}

		hipDeviceSynchronize();
		++iteration;
	}

	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);

	// PRINT TIME ELAPSED
	printf("Time elapsed: %f\n", t.elapsed());

	// Pull rod into host for testing
	floatp* h_rod = (floatp *)malloc(size);
	cudaStatus = hipMemcpy(h_rod, d_rod, size, hipMemcpyDeviceToHost);
	CHECK(cudaStatus);

	for (int i = 0; i <= 10; i += 1)
	{
		printf("%f\t", h_rod[i]);
	}
	printf("\n");
	printf("%f\n", h_rod[H_NUM_SLICES - 1]);

	int ind = 0;
	while (abs(h_rod[ind] - 23.00) > 0.0001)
	{
		++ind;
	}
	printf("first untouched = %d\n", ind);

	// Free memory
	hipFree(d_rod);
	hipFree(d_rod2);
	free(h_rod);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	CHECK(hipDeviceReset());
}

void Heat2D()
{
	hipError_t cudaStatus;
	size_t size = H_ROOM_X * H_ROOM_Y * sizeof(floatp);

	// START TIMER
	boost::timer t;

	// Allocate space for rooms
	floatp* d_room = NULL;
	cudaStatus = hipMalloc((void**)&d_room, size);
	CHECK(cudaStatus);

	floatp* d_room2 = NULL;
	cudaStatus = hipMalloc((void**)&d_room2, size); // TEST WITH PADDING
	CHECK(cudaStatus);

	// Calculate threads/blocks for initialize kernel
	dim3 blockDims(32, 32, 1);

	int xBlocks = H_ROOM_X / 32;
	if (H_ROOM_X % 32 != 0)
		++xBlocks;
	int yBlocks = H_ROOM_Y / 32;
	if (H_ROOM_Y % 32 != 0)
		++yBlocks;
	dim3 gridDims(xBlocks, yBlocks, 1);

	// Perform initialize kernel
	printf("CUDA 'intitialize2D' kernel launch with %d x %d blocks of %d x %d threads\n",
		gridDims.x, gridDims.y, blockDims.x, blockDims.y);

	initialize2D <<< gridDims, blockDims >>> (d_room);
	initialize2D <<< gridDims, blockDims >>> (d_room2);

	cudaStatus = hipGetLastError();
	CHECK(cudaStatus);

	// Perform applyHeat kernel for a specified number of iterations (time steps)
	printf("CUDA 'applyHeat2D' kernel launch with %d x %d blocks of %d x %d threads\n", 
		gridDims.x, gridDims.y, blockDims.x, blockDims.y);

	// Save initial image
	floatp* h_room = (floatp *)malloc(size);
	cudaStatus = hipMemcpy(h_room, d_room, size, hipMemcpyDeviceToHost);
	CHECK(cudaStatus);
	DrawImage(h_room, H_ROOM_Y, H_ROOM_X, 0);

	// Divide up iterations
	for (int step = 0; step != STEPS; ++step)
	{ 
		int iteration = 0;
		while (iteration < ITERATIONS / STEPS)
		{
			if (iteration % 2 == 0)
			{
				applyHeat2D <<< gridDims, blockDims >>> (d_room, d_room2);
			}
			else
			{
				applyHeat2D <<< gridDims, blockDims >>> (d_room2, d_room);
			}

			hipDeviceSynchronize();
			++iteration;
		}
		cudaStatus = hipGetLastError();
		CHECK(cudaStatus);

		// Draw image
		if (DRAW_IMG)
		{ 
			h_room = (floatp *)malloc(size);
			cudaStatus = hipMemcpy(h_room, d_room, size, hipMemcpyDeviceToHost);
			CHECK(cudaStatus);
			DrawImage(h_room, H_ROOM_Y, H_ROOM_X, step+1);
		}
	}

	// PRINT TIME ELAPSED
	printf("Time elapsed: %f\n", t.elapsed());

	// Free memory
	hipFree(d_room);
	hipFree(d_room2);
	free(h_room);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	CHECK(hipDeviceReset());
}



int main(int argc, char* argv[])
{ 
	//OneBlockHeat();
	MultiBlockHeat();
	//Heat2D();

    return 0;
}

